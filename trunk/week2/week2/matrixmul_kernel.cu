#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * This software and the information contained herein is PROPRIETARY and 
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and 
 * conditions of a Non-Disclosure Agreement.  Any reproduction or 
 * disclosure to any third party without the express written consent of 
 * NVIDIA is prohibited.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* Matrix multiplication: C = A * B.
 * Device code.
 */
#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include <stdio.h>
#include "matrixmul.h"


/*  save a few registers  */
#define TX (threadIdx.x)
#define TY (threadIdx.y)
#define BX (blockIdx.x)
#define BY (blockIdx.y)
#define W (M.width)


/* In general se pp 35 + in CUDA best practices (pedersen) */

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
// Matrix multiplication kernel thread specification
__global__ void MatrixMulKernel(Matrix M, Matrix N, Matrix P)
{

  int row = BY * blockDim.y + TY;
  int col = BX * blockDim.x + TX;

  /* we shouldn't calculate these */
  if (row >= P.height || col >= P.width)
    return;

  float pvalue = 0;	
  for(int k = 0; k < M.width ; ++k) {
    pvalue += M.elements[row*M.width + k] * N.elements[k*N.width + col];
  }
  P.elements[row*P.width + col] = pvalue;

}

// Matrix multiplication kernel thread specification
__global__ void MatrixMulKernelTiled(Matrix M, Matrix N, Matrix P)
{
   __shared__ float Ms[TILE_WIDTH][TILE_WIDTH];
   __shared__ float Ns[TILE_WIDTH][TILE_WIDTH];

   int row = BY * TILE_WIDTH + TY;
   int col = BX * TILE_WIDTH + TX;

   float pvalue = 0;	
   for(int m = 0; m < (int) ceil( 1.0f * W/TILE_WIDTH); ++m) {

      /* loading  with inline zero padding??*/
     if (row >= P.height || col >= P.width) {
         Ns[TY][TX] =  Ms[TY][TX] = 0.0f;
      } else {
         Ms[TY][TX] = M.elements[row * W + (m*TILE_WIDTH + TX)];
         Ns[TY][TX] = N.elements[(m*TILE_WIDTH + TY) * W + col];
      }
      __syncthreads();
      for(int k = 0; k < TILE_WIDTH; ++k) {
         pvalue += Ms[TY][k] * Ns[k][TX];
      }
      __syncthreads();

   }
   if (row >= P.height || col >= P.width) return;
   P.elements[row*P.width + col] = pvalue;
}

texture <float, 1> Mtex;
texture <float, 1> Ntex;

// Matrix multiplication kernel thread specification
__global__ void MatrixMulKernelTextured(Matrix M, Matrix N, Matrix P)
{



  int row = BY * blockDim.y + TY;
  int col = BX * blockDim.x + TX;

  /* we shouldn't calculate these */
  if (row >= P.height || col >= P.width)
    return;

  float pvalue = 0;	
  for(int k = 0; k < M.width ; ++k) {
    pvalue += tex1Dfetch(Mtex, (float) (row*M.width + k))
      * tex1Dfetch(Ntex, (float) (k*N.width + col));
  }
  P.elements[row*P.width + col] = pvalue;
}


#endif // #ifndef _MATRIXMUL_KERNEL_H_
