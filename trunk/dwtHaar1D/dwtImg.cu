#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <cutil_inline.h>

#include "dwtImg.h"
#include "dwtHaar1D.h"
#include "dwtHaar1D_kernel.h"



#define HANDLE_ERROR(X, MSG) do { if (hipSuccess != X)			\
      {printf("cuda fejl: %s\n i %s\n", hipGetErrorString(hipGetLastError()), MSG); \
	exit(42);} } while (0)



__device__
void decompositionStep(float *C, float *out, int half_step) {
  unsigned int id = threadIdx.x;
  int offset;
  float data0; 
  float data1;
  
  
  if (id < half_step) {
    offset = blockIdx.y * 1024 ;
    data0 = C[offset + 2 * id]; 
    data1 = C[offset + 2 * id + 1];
  }
  __syncthreads(); 
  
  if (id < half_step) {
    out[offset + id] =             (data0 + data1) * INV_SQRT_2;
    out[offset + id + half_step] = (data0 - data1) * INV_SQRT_2;
  }
  __syncthreads(); 
}



__device__
void recomposeStep(float *D, float *out, int half_step, int level) {
  unsigned int id = threadIdx.x;
  if (id < half_step) {
    int offset = blockIdx.y * 1024;
    float coarse = D[offset + id]; 
    float detail = D[offset + id + half_step];
    out[offset + 2 * id] = (coarse + detail )*INV_SQRT_2;
    out[offset + 2 * id + 1] = (coarse - detail )*INV_SQRT_2;
  } 

  __syncthreads();

}


__global__
void recompose(float *D, float * out, int levels) {
  unsigned int half_step = 2 * blockDim.x>>levels;
  unsigned int id = threadIdx.x;
  int offset = blockIdx.y * 1024 ;
  for (int i=0; i < levels; i++) {
    recomposeStep(D, out, half_step, i);
    half_step <<= 1;
    D = out;    
  }
}
  



__global__
void decomposition(float *C, float * out, int levels) {
  unsigned int id = threadIdx.x;
  unsigned int half_step = blockDim.x;
  int offset = blockIdx.y * 1024 ;
  /* normalize dataset */
  C[offset + id] *= INV_SQRT_2;
  C[offset + id + half_step] *= INV_SQRT_2;
  
  __syncthreads();
  
  for (int i=0; i < levels; i++) {
    decompositionStep(C, out, half_step);
    half_step >>= 1; 		/* div 2 */
    C = out;
  }
}


/* something quick and dirty */
void test_img(const char* file, int levels) {
 
  /*   showIds(2, 4); */

  /*   return; */


  unsigned char *data;
  unsigned int img_w, img_h;
  cutLoadPPM4ub(file, &data, &img_w, &img_h);
  printf("img_w= %d, img_h= %d\n", img_w, img_h);


  // allocate device mem

  const unsigned int smem_size = sizeof(int) * img_w * img_h;
  const unsigned int img_smem_size = sizeof(float) * img_w * img_h;
  const unsigned int slength =  img_w * img_h;

  printf("memory use sizeof(float) = %d, sizeof(int) = %d\n", sizeof(float), sizeof(int));

  int *int_image;
  float *img_data;
  int_image = NULL;

  cutilSafeCall( hipMalloc( (void**) &int_image, smem_size));
  cutilSafeCall( hipMalloc( (void**) &img_data, img_smem_size));

  // copy input data to device
  cutilSafeCall( hipMemcpy( int_image, data, smem_size, hipMemcpyHostToDevice) );


  // device out data
  float* d_odata = NULL;

  cutilSafeCall( hipMalloc( (void**) &d_odata, img_smem_size));

  
  // clear result memory
  float* tmp = (float*) malloc( img_smem_size);
  for( unsigned int i = 0; i < slength; ++i) {
    tmp[i] = 0.0;
  }
  cutilSafeCall( hipMemcpy( d_odata, tmp, smem_size, 
			     hipMemcpyHostToDevice) ); 
  free( tmp);



  //to grayscale
  dim3 grid, block;
  block.x = 512;
  grid.x = (img_h *img_w)/block.x;

  to_grayscale_floats<<< grid, block>>>(int_image, img_data, img_w * img_h);
  from_grayscale_floats_to_ppm("original.ppm", img_data, img_w, img_h);


  // 2D signal so the arrangement of elements is also 2D
  dim3  block_size;
  dim3  grid_size;  

  block_size.x = 512;
  grid_size.x = 1; 		/* one row */
  grid_size.y = 1024;		/* corresponding to cols */

  // run kernel
  decomposition<<<grid_size, block_size >>>( img_data, d_odata, levels);  


  //from grayscale
  from_grayscale_floats_to_ppm("output.ppm", d_odata, img_w, img_h);

  /* recompose */
  cutilSafeCall( hipMemcpy(img_data, d_odata, smem_size, hipMemcpyDeviceToDevice) );
  recompose<<<grid_size, block_size >>>( img_data, d_odata, levels);  

  //from grayscale
  from_grayscale_floats_to_ppm("recomposed.ppm", d_odata, img_w, img_h);





}




__global__
void to_grayscale_floats(int *in, float *out, int size) {
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= size)
    return;

  int pixel = in[tid];
  unsigned char r = (pixel >> 16) & 0xFF;
  unsigned char g = (pixel >> 8) & 0xFF;
  unsigned char b = pixel & 0xFF;

  out[tid] = (r + g + b)/3.0f;

}


__global__
void from_grayscale_floats(float *in, int* out, int size) {
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= size)
    return;
  float p = in[tid];
  p = max(min(255.0f, p), 0.0f);
  unsigned char val = (unsigned char) p;
  int pixel = val << 16 | val << 8 | val;
  out[tid] = pixel;

}



__global__
void simple_copy_kernel(int *in, int *out) {
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  out[tid] = in[(tid%1025)];
}


void from_grayscale_floats_to_ppm(const char *filename, float *d_odata, int img_w, int img_h) {

  int img_size = (img_h *img_w);
  int mem_size = img_size*sizeof(int);
  dim3 block, grid;

  block.x = 512;
  grid.x = img_size/block.x;
 


  int *int_image, *data;
  cutilSafeCall( hipMalloc( (void**) &int_image, mem_size));
  data = (int*) malloc(sizeof(int) * img_size);

  from_grayscale_floats<<< grid, block>>>(d_odata, int_image ,img_w * img_h);
  HANDLE_ERROR(hipPeekAtLastError(), "after from grayscale");


  /* copy result back */
  cutilSafeCall( hipMemcpy(data, int_image, mem_size , hipMemcpyDeviceToHost) );

  /* write result out */
  cutSavePPM4ub(filename, (unsigned char*) data, img_w, img_h);

  /* free stuff */
  free(data);
  cutilSafeCall( hipFree(int_image));

}

__global__ void 
dwtHaar2D( float* id, float* od, float* approx_final, 
	   const unsigned int dlevels,
	   const unsigned int slength_step_half,
	   const int bdim ) 
{ 
  // shared memory for part of the signal
  extern __shared__ float shared[];  

  // thread runtime environment, 1D parametrization
  const int gdim = gridDim.x;
  // const int bdim = blockDim.x;
  const int bid = blockIdx.x;
  const int tid = threadIdx.x;  

  const int col_offset = 1024 * blockIdx.y;


  // global thread id (w.r.t. to total data set)
  const int tid_global = (bid * bdim) + tid;    
  unsigned int idata = (bid * (2 * bdim)) + tid;

  // read data from global memory
  shared[tid] = id[idata +col_offset];
  shared[tid + bdim] = id[idata + bdim + col_offset];
  __syncthreads();

  // this operation has a two way bank conflicts for all threads, this are two
  // additional cycles for each warp -- all alternatives to avoid this bank
  // conflict are more expensive than the one cycle introduced by serialization
  float data0 = shared[2*tid];
  float data1 = shared[(2*tid) + 1];
  __syncthreads();

  // detail coefficient, not further referenced so directly store in
  // global memory
  od[tid_global + slength_step_half + col_offset] = (data0 - data1) * INV_SQRT_2;

  // offset to avoid bank conflicts
  // see the scan example for a more detailed description
  unsigned int atid = tid + (tid >> LOG_NUM_BANKS);

  // approximation coefficient
  // store in shared memory for further decomposition steps in this global step
  shared[atid] = (data0 + data1) * INV_SQRT_2;

  // all threads have to write approximation coefficient to shared memory before 
  // next steps can take place
  __syncthreads();

  // early out if possible
  // the compiler removes this part from the source because dlevels is 
  // a constant shader input
  // note: syncthreads in bodies of branches can lead to dead-locks unless the
  // the condition evaluates the same way for ALL threads of a block, as in 
  // this case
  if( dlevels > 1) 
    {
      // offset to second element in shared element which has to be used for the 
      // decomposition, effectively 2^(i - 1)
      unsigned int offset_neighbor = 1;
      // number of active threads per decomposition level
      // identiacal to the offset for the detail coefficients
      unsigned int num_threads = bdim >> 1;

      // index for the first element of the pair to process
      // the representation is still compact (and therefore still tid * 2) 
      // because the first step operated on registers and only the result has been
      // written to shared memory
      unsigned int idata0 = tid * 2;

      // offset levels to make the loop more efficient
      for( unsigned int i = 1; i < dlevels; ++i) 
        {
	  // Non-coalesced writes occur if the number of active threads becomes 
	  // less than 16 for a block because the start address for the first 
	  // block is not always aligned with 64 byte which is necessary for 
	  // coalesced access. However, the problem only occurs at high levels 
	  // with only a small number of active threads so that the total number of 
	  // non-coalesced access is rather small and does not justify the 
	  // computations which are necessary to avoid these uncoalesced writes
	  // (this has been tested and verified)
	  if( tid < num_threads) 
            {
	      // update stride, with each decomposition level the stride grows by a 
	      // factor of 2
	      unsigned int idata1 = idata0 + offset_neighbor;

	      // position of write into global memory
	      unsigned int g_wpos = (num_threads * gdim) + (bid * num_threads) + tid +  col_offset;

	      // compute wavelet decomposition step

	      // offset to avoid bank conflicts
	      unsigned int c_idata0 = idata0 + (idata0 >> LOG_NUM_BANKS);
	      unsigned int c_idata1 = idata1 + (idata1 >> LOG_NUM_BANKS);

	      // detail coefficient, not further modified so directly store 
	      // in global memory
	      od[g_wpos] = (shared[c_idata0] - shared[c_idata1]) * INV_SQRT_2;

	      // approximation coefficient
	      // note that the representation in shared memory becomes rather sparse 
	      // (with a lot of holes inbetween) but the storing scheme in global 
	      // memory guarantees that the common representation (approx, detail_0, 
	      // detail_1, ...)
	      // is achieved
	      shared[c_idata0] = (shared[c_idata0] + shared[c_idata1]) * INV_SQRT_2;

	      // update storage offset for details
	      num_threads = num_threads >> 1;   // div 2
	      offset_neighbor <<= 1;   // mul 2 
	      idata0 = idata0 << 1;   // mul 2     
            }

	  // sync after each decomposition step
	  __syncthreads();
        }

      // write the top most level element for the next decomposition steps
      // which are performed after an interblock syncronization on host side
      if( 0 == tid) 
        {
	  approx_final[bid + col_offset] = shared[0];
        }

    } // end early out if possible
}

