#include "hip/hip_runtime.h"



// clamp x to range [a, b]
__device__ float clamp(float x, float a, float b)
{
  return max(a, min(b, x));
}

__device__ int clamp(int x, int a, int b)
{
  return max(a, min(b, x));
}

// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b)
{
  r = clamp(r, 0.0f, 255.0f);
  g = clamp(g, 0.0f, 255.0f);
  b = clamp(b, 0.0f, 255.0f);
  return (int(b)<<16) | (int(g)<<8) | int(r);
}

// get pixel from 2D image, with clamping to border

__device__ int getPixel(int *data, int x, int y, int width, int height)
{
  x = clamp(x, 0, width-1);
  y = clamp(y, 0, height-1);

  return data[y*width+x];
}

__global__ void
cudaProcess(int* g_data, int* g_odata, int imgw, int imgh, float * device_stencil_data, int stencil_width, int stencil_height)
{
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int bw = blockDim.x;
  int bh = blockDim.y;
  int x = blockIdx.x*bw + tx;
  int y = blockIdx.y*bh + ty;

  // perform convolution
  float rsum = 0.0;
  float gsum = 0.0;
  float bsum = 0.0;

  for(int dy=0; dy<stencil_height; dy++) {
    for(int dx=0; dx<stencil_width; dx++) {

      int pixel = getPixel(g_data, x+dx-(stencil_width-1)/2, y+dy-(stencil_height-1)/2, imgw, imgh);

      float r = float(pixel&0xff);
      float g = float((pixel>>8)&0xff);
      float b = float((pixel>>16)&0xff);

      float stencil_value = device_stencil_data[dx+dy*stencil_width];

      rsum += r*stencil_value;
      gsum += g*stencil_value;
      bsum += b*stencil_value;
    }
  }

  g_odata[y*imgw+x] = rgbToInt(rsum, gsum, bsum);
}

__global__ void
cudaProcessEx3(int* g_data, int* g_odata, int imgw, int imgh, float * device_stencil_data)
{
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int bw = blockDim.x;
  int bh = blockDim.y;
  int x = blockIdx.x*bw + tx;
  int y = blockIdx.y*bh + ty;

  // perform convolution
  float rsum = 0.0;
  float gsum = 0.0;
  float bsum = 0.0;

  /* sharemem for stencil */
  __shared__ float s_stencil[STENCIL_HEIGHT][STENCIL_WIDTH];


  if (tx < STENCIL_WIDTH && ty < STENCIL_HEIGHT )
    s_stencil[ty][tx] = device_stencil_data[ty*STENCIL_WIDTH + tx];
  __syncthreads();

  for(int dy=0; dy<STENCIL_HEIGHT; dy++) {
    for(int dx=0; dx<STENCIL_WIDTH; dx++) {

      int pixel = getPixel(g_data, x+dx-(STENCIL_WIDTH-1)/2, y+dy-(STENCIL_HEIGHT-1)/2, imgw, imgh);

      float r = float(pixel&0xff);
      float g = float((pixel>>8)&0xff);
      float b = float((pixel>>16)&0xff);

      float stencil_value = s_stencil[dy][dx];

      rsum += r*stencil_value;
      gsum += g*stencil_value;
      bsum += b*stencil_value;
    }
  }

  g_odata[y*imgw+x] = rgbToInt(rsum, gsum, bsum);
}

#define CLAMP(X, MIN, MAX) ( (X > MAX) ? MAX : (X < MIN) ? MIN : X )

__global__ void
cudaProcessEx4(int* g_data, int* g_odata, int imgw, int imgh, float * device_stencil_data)
{
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int x = blockIdx.x*BLOCK_WIDTH + tx;
  int y = blockIdx.y*BLOCK_HEIGHT + ty;

  // perform convolution
  float rsum = 0.0;
  float gsum = 0.0;
  float bsum = 0.0;

  /* sharemem for stencil */
  __shared__ float s_stencil[STENCIL_HEIGHT][STENCIL_WIDTH];
  __shared__ int tile[(2+BLOCK_HEIGHT)][(2+BLOCK_WIDTH)];

  if (tx < STENCIL_WIDTH && ty < STENCIL_HEIGHT )
    s_stencil[ty][tx] = device_stencil_data[ty*STENCIL_WIDTH + tx];
  /* read ze tile */
  
  /* x :  CLAMP( blockIdx.x * blockDim.x + (tx-1), 0, imgw)  */
  /* y :  CLAMP( blockIdx.y * blockDim.y + (ty-1), 0, imgh)  */
  tile[ty][tx] = g_data[ (y-1) * imgw + x-1];


  if (ty < 2) 
    tile[ty+BLOCK_HEIGHT][BLOCK_WIDTH] = g_data[(y+2) *imgw + x];
  if (tx < 2) 
    tile[BLOCK_HEIGHT][tx+BLOCK_WIDTH] = g_data[ y *imgw + x+2];

  __syncthreads();
    



  for(int dy=-(STENCIL_HEIGHT-1)/2 ; dy <= (STENCIL_HEIGHT-1)/2; dy++) {
    for(int dx=-(STENCIL_WIDTH-1)/2; dx <= (STENCIL_WIDTH-1)/2; dx++) {

      int pixel = tile[ty+dy+1][tx+dx+1];

      float r = float(pixel&0xff);
      float g = float((pixel>>8)&0xff);
      float b = float((pixel>>16)&0xff);

      float stencil_value =   (dy==0 && dx==0) ? 1.0f : 0.0f; // s_stencil[dy][dx];

      rsum += r*stencil_value;
      gsum += g*stencil_value;
      bsum += b*stencil_value;
    }
  }

  g_odata[y*imgw+x] = rgbToInt(rsum, gsum, bsum);
}
